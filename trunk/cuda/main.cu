#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <float.h>
#include <GL/glew.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <GL/glut.h>

#define IMAGE			"bigbrain.bmp"

#define ITERATIONS   5000
#define THRESHOLD	 100
#define EPSILON		 40

#define RITS		 50

float *phi, *D;
uchar4 *h_Src, *h_Mask;
int imageW, imageH, N;

float *d_phi, *d_D;


void LoadBMPFile(uchar4 **dst, int *width, int *height, const char *name);
void sedt2d(int *_d,unsigned char *_bimg,int _h,int _w);


int its=0;

int r;
int c;
int i;

__global__ void updatephi( float *d_phi, float *d_phi1, float *d_D, int imageW, int imageH);

void init_phi(){

	int *init;
	unsigned char *mask;
	const char *mask_path = "mask.bmp";
	if((init=(int *)malloc(imageW*imageH*sizeof(int)))==NULL)printf("ME_INIT\n");
	if((phi=(float *)malloc(imageW*imageH*sizeof(float)))==NULL)printf("ME_PHI\n");
	mask = (unsigned char *)malloc(imageW*imageH*sizeof(unsigned char));

	//printf("Init Mask\n");
	LoadBMPFile(&h_Mask, &imageW, &imageH, mask_path);
	

	for(r=0;r<imageH;r++){
		for(c=0;c<imageW;c++){
			mask[r*imageW+c] = (h_Mask[r*imageW+c].x)/255;
			//printf("%3d ", mask[r*imageW+c]);
		}
		//printf("\n");
	}

	sedt2d(init,mask,imageH,imageW);

	//printf("sdf of init mask\n");
	for(r=0;r<imageH;r++){
		for(c=0;c<imageW;c++){
			phi[r*imageW+c]=(float)init[r*imageW+c];
			if(phi[r*imageW+c]>0){
				phi[r*imageW+c]=0.5*sqrt(abs(phi[r*imageW+c]));
			} else {
				phi[r*imageW+c]=-0.5*sqrt(abs(phi[r*imageW+c]));
			}
			//printf("%6.3f ", phi[r*imageW+c]);
		}
		//printf("\n");
	}

	free(init);
	free(mask);
}



void cuda_update(){

    float *d_phi1;
	
	hipMalloc((void**)&d_phi1,         sizeof(float)*imageW*imageH);

	hipMemcpy(d_phi1, phi, sizeof(float)*imageW*imageH, hipMemcpyHostToDevice);


	dim3 dimGrid( ((imageW-1)/4) + 1, ((imageH-1)/32) +1 );
	dim3 dimBlock(4, 32);


	updatephi<<< dimGrid, dimBlock>>>(d_phi, d_phi1, d_D,  imageW, imageH);

	hipMemcpy(phi, d_phi, sizeof(float)*imageW*imageH, hipMemcpyDeviceToHost);


	printf("%3d\n", its);
	//printf("Speed Function\n");	
	//for(int r=0;r<imageH;r++){
	//	for(int c=0;c<imageW;c++){
	//		printf("%4.1f ", phi[r*imageW+c]);
	//	}
	//	printf("\n");
	//}


	hipFree(d_phi1);

}

void disp(void){
	
	glClear(GL_COLOR_BUFFER_BIT);

	cuda_update();

	glDrawPixels(imageW, imageH, GL_GREEN, GL_FLOAT, phi);

	glutSwapBuffers();
	
	its++;

	if(its<ITERATIONS){
		glutPostRedisplay();
	} else {

		}
}

int main(int argc, char** argv){

	const char *image_path = IMAGE;
	
	//TODO : declare ALL variables here

	LoadBMPFile(&h_Src, &imageW, &imageH, image_path);
	D = (float *)malloc(imageW*imageH*sizeof(float));

	//printf("Input Image\n");
	for(r=0;r<imageH;r++){
		for(c=0;c<imageW;c++){
			D[r*imageW+c] = h_Src[r*imageW+c].x;
			/*printf("%3.0f ", D[r*imageW+c]);*/
		}
		//printf("\n");
	}

	N = imageW*imageH;

	for(i=0;i<N;i++){
		D[i] = EPSILON - abs(D[i] - THRESHOLD);
	}

	init_phi();



	hipMalloc((void**)&d_D,         sizeof(float)*imageW*imageH);
    hipMemcpy(d_D, D, sizeof(float)*imageW*imageH, hipMemcpyHostToDevice);

	hipMalloc((void**)&d_phi, sizeof(float)*imageW*imageH);
	

		  // GL initialisation
		  glutInit(&argc, argv);
		  glutInitDisplayMode(GLUT_ALPHA | GLUT_DOUBLE);
		  glutInitWindowSize(imageW,imageH);
		  glutInitWindowPosition(100,100);
		  glutCreateWindow("GL Level Set Evolution");
		  glClearColor(0.0,0.0,0.0,0.0);


		  glutDisplayFunc(disp);
		  glutMainLoop();
	
}




//TODO Memory Malloc Free

//TODO Timer

//TODO Comment Code
