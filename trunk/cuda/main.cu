#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <float.h>
#include <GL/glew.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <GL/glut.h>

#define IMAGE			"squares.bmp"
#define IMAGE_FILE_NAME "Output.pgm"


#define ITERATIONS   500
#define THRESHOLD	 10
#define EPSILON		 100



#define RITS		 50

float *phi, *D;
uchar4 *h_Src, *h_Mask;
int imageW, imageH, N;

float *d_phi, *d_D;


void LoadBMPFile(uchar4 **dst, int *width, int *height, const char *name);
void sedt2d(int *_d,unsigned char *_bimg,int _h,int _w);


int its=0;

int r;
int c;
int i;

__global__ void updatephi( float *d_phi, float *d_phi1, float *d_D, int imageW, int imageH, 
						 float *dx, float *dxplus, float *dxminus, float *dxplusy, float *dxminusy,
						 float *maxdxplus, float *maxminusdxminus, float *mindxplus, float *minminusdxminus,
						 float *dy, float *dyplus, float *dyminus, float *dyplusx, float *dyminusx,
						 float *maxdyplus, float *maxminusdyminus, float *mindyplus, float *minminusdyminus,
						 float *nplusx,float *nplusy,float *nminusx,float *nminusy,float *curvature,
						 float *gradphimax,float *gradphimin, float *F, float *gradphi);

void init(){

	int *initsdf;
	unsigned char *mask;
	const char *mask_path = "mask.bmp";
	const char *image_path = IMAGE;
	if((initsdf=(int *)malloc(imageW*imageH*sizeof(int)))==NULL)printf("ME_INITSDF\n");
	if((phi=(float *)malloc(imageW*imageH*sizeof(float)))==NULL)printf("ME_PHI\n");
	mask = (unsigned char *)malloc(imageW*imageH*sizeof(unsigned char));
	D = (float *)malloc(imageW*imageH*sizeof(float));

	//printf("Init Mask\n");
	LoadBMPFile(&h_Mask, &imageW, &imageH, mask_path);
	LoadBMPFile(&h_Src, &imageW, &imageH, image_path);

	N = imageW*imageH;

	for(r=0;r<imageH;r++){
		for(c=0;c<imageW;c++){
			mask[r*imageW+c] = (h_Mask[r*imageW+c].x)/255;
			//printf("%3d ", mask[r*imageW+c]);
		}
		//printf("\n");
	}

	sedt2d(initsdf,mask,imageH,imageW);
	
	
	//printf("sdf of init mask\n");
	for(i=0;i<N;i++){
		phi[i]=(float)initsdf[i];
		if(phi[i]>0){
			phi[i]=0.5*sqrt(abs(phi[i]));
		} else {
			phi[i]=-0.5*sqrt(abs(phi[i]));
		}
	}

		for(i=0;i<N;i++){
			D[i] =(float)h_Src[i].x;
			D[i] = EPSILON - abs(D[i] - THRESHOLD);
	}

	free(init);
	free(mask);


}

//void reinit_phi(){
//
//	int *intphi;
//	unsigned char *reinit;
//	if((intphi=(int *)malloc(imageW*imageH*sizeof(int)))==NULL)printf("ME_INIT\n");
//	reinit=(unsigned char *)malloc(imageW*imageH*sizeof(unsigned char));//TODO check
//
//	for(i=0;i<N;i++){
//		if(phi[i]<0){
//			phi[i]=1;
//		} else {
//			phi[i]=0;
//		}
//		reinit[i]=(int)phi[i];
//	}
//
//
//	sedt2d(intphi,reinit,imageH,imageW);
//
//	printf("ReInit @ %4d its\n",its);
//	for(r=0;r<imageH;r++){
//		for(c=0;c<imageW;c++){
//			phi[r*imageW+c]=(float)intphi[r*imageW+c];
//			if(phi[r*imageW+c]>0){
//				phi[r*imageW+c]=0.5*sqrt(abs(phi[r*imageW+c]));
//			} else {
//				phi[r*imageW+c]=-0.5*sqrt(abs(phi[r*imageW+c]));
//			}
//			//printf("%6.3f ", phi[r*imageW+c]);
//		}
//		//printf("\n");
//	}
//
//	free(reinit);
//	free(intphi);
//}


void cuda_update(dim3 dimGrid, dim3 dimBlock){

	float *dx,*dxplus,*dxminus,*dxplusy,*dxminusy;
	float *maxdxplus, *maxminusdxminus, *mindxplus, *minminusdxminus;

	float *dy, *dyplus,*dyminus,*dyplusx,*dyminusx;
	float *maxdyplus, *maxminusdyminus, *mindyplus, *minminusdyminus;

	float *gradphimax, *gradphimin, *nplusx, *nplusy, *nminusx, *nminusy, *curvature;
    float *F, *gradphi, *d_phi1;
	
	hipMalloc((void**)&dx,         sizeof(float)*imageW*imageH);
	hipMalloc((void**)&dxplus,     sizeof(float)*imageW*imageH);
	hipMalloc((void**)&dxminus,    sizeof(float)*imageW*imageH);
	hipMalloc((void**)&dxplusy,    sizeof(float)*imageW*imageH);
	hipMalloc((void**)&dxminusy,    sizeof(float)*imageW*imageH);
	hipMalloc((void**)&maxdxplus,    sizeof(float)*imageW*imageH);
	hipMalloc((void**)&maxminusdxminus,    sizeof(float)*imageW*imageH);
	hipMalloc((void**)&mindxplus,    sizeof(float)*imageW*imageH);
	hipMalloc((void**)&minminusdxminus,    sizeof(float)*imageW*imageH);

	hipMalloc((void**)&dy,         sizeof(float)*imageW*imageH);
	hipMalloc((void**)&dyplus,     sizeof(float)*imageW*imageH);
	hipMalloc((void**)&dyminus,    sizeof(float)*imageW*imageH);
	hipMalloc((void**)&dyplusx,    sizeof(float)*imageW*imageH);
	hipMalloc((void**)&dyminusx,    sizeof(float)*imageW*imageH);
	hipMalloc((void**)&maxdyplus,    sizeof(float)*imageW*imageH);
	hipMalloc((void**)&maxminusdyminus,    sizeof(float)*imageW*imageH);
	hipMalloc((void**)&mindyplus,    sizeof(float)*imageW*imageH);
	hipMalloc((void**)&minminusdyminus,    sizeof(float)*imageW*imageH);

	hipMalloc((void**)&gradphimax,         sizeof(float)*imageW*imageH);
	hipMalloc((void**)&gradphimin,         sizeof(float)*imageW*imageH);
	hipMalloc((void**)&nplusx,         sizeof(float)*imageW*imageH);
	hipMalloc((void**)&nplusy,         sizeof(float)*imageW*imageH);
	hipMalloc((void**)&nminusx,         sizeof(float)*imageW*imageH);
	hipMalloc((void**)&nminusy,         sizeof(float)*imageW*imageH);
	hipMalloc((void**)&curvature,         sizeof(float)*imageW*imageH);

	hipMalloc((void**)&F,         sizeof(float)*imageW*imageH);
	hipMalloc((void**)&gradphi,         sizeof(float)*imageW*imageH);
	hipMalloc((void**)&d_phi1,         sizeof(float)*imageW*imageH);

	hipMemcpy(d_phi1, phi, sizeof(float)*imageW*imageH, hipMemcpyHostToDevice);




	updatephi<<< dimGrid, dimBlock>>>( d_phi, d_phi1, d_D,  imageW,  imageH, 
						 dx, dxplus, dxminus, dxplusy, dxminusy,
						 maxdxplus, maxminusdxminus, mindxplus, minminusdxminus,
						 dy, dyplus, dyminus, dyplusx, dyminusx,
						 maxdyplus, maxminusdyminus, mindyplus, minminusdyminus,
						 nplusx,nplusy,nminusx,nminusy,curvature,
						 gradphimax,gradphimin, F, gradphi);

	hipMemcpy(phi, d_phi, sizeof(float)*imageW*imageH, hipMemcpyDeviceToHost);

	//printf("Speed Function\n");	
	//for(int r=0;r<imageH;r++){
	//	for(int c=0;c<imageW;c++){
	//		printf("%3.0f ", phi[r*imageW+c]);
	//	}
	//	printf("\n");
	//}

	hipFree(dx);
	hipFree(dxplus);
	hipFree(dxminus);
	hipFree(dxplusy);
	hipFree(dxminusy);
	hipFree(maxdxplus);
	hipFree(maxminusdxminus);
	hipFree(mindxplus);
	hipFree(mindxplus);
	hipFree(dy);
	hipFree(dyplus);
	hipFree(dyminus);
	hipFree(dyplusx);
	hipFree(dyminusx);
	hipFree(maxdyplus);
	hipFree(maxminusdyminus);
	hipFree(mindyplus);
	hipFree(minminusdyminus);
	hipFree(gradphimax);
	hipFree(gradphimin);
	hipFree(nplusx);
	hipFree(nplusy);
	hipFree(nminusx);
	hipFree(nminusy);
	hipFree(curvature);
	hipFree(F);
	hipFree(gradphi);
	hipFree(d_phi1);

}



int main(int argc, char** argv){

	init();

	dim3 dimGrid( ((imageW-1)/4) + 1, ((imageH-1)/32) +1 );
	dim3 dimBlock(4, 32);

	hipMalloc((void**)&d_D,         sizeof(float)*imageW*imageH);
    hipMemcpy(d_D, D, sizeof(float)*imageW*imageH, hipMemcpyHostToDevice);

	hipMalloc((void**)&d_phi, sizeof(float)*imageW*imageH);
	

	for(its=0;its<ITERATIONS;its++){
		cuda_update(dimGrid, dimBlock);
	}
	
}




//TODO Memory Malloc Free

//TODO Timer

//TODO Comment Code
