
#include <hip/hip_runtime.h>

#define ALPHA		 0.03
#define DT			 0.2

__global__ void updatephi( float *d_phi, float *d_phi1, float *d_D, int imageW, int imageH)

{
	int c= blockIdx.x * blockDim.x + threadIdx.x;
	int r= blockIdx.y * blockDim.y + threadIdx.y;
	int ind= r*imageW+c;

	if(ind<imageW*imageH){

		float dx,dxplus,dxminus,dxplusy,dxminusy;
		float maxdxplus, maxminusdxminus, mindxplus, minminusdxminus;

		float dy, dyplus,dyminus,dyplusx,dyminusx;
		float maxdyplus, maxminusdyminus, mindyplus, minminusdyminus;

		float gradphimax, gradphimin, nplusx, nplusy, nminusx, nminusy, curvature;
		float F, gradphi;

		if(c==0||c==imageW-1){dx=0;} else {dx=(d_phi1[ind+1]-d_phi1[ind-1])/2;}
		if(c==imageW-1){dxplus=0;} else {dxplus=(d_phi1[ind+1]-d_phi1[ind]);}
		if(c==0){dxminus=0;} else {dxminus=(d_phi1[ind]-d_phi1[ind-1]);}
		if(r==0||c==0||c==imageW-1){dxplusy=0;} else {dxplusy=(d_phi1[ind-imageW+1]-d_phi1[ind-imageW-1])/2;}
		if(r==imageH-1||c==0||c==imageW-1){dxminusy=0;} else {dxminusy=(d_phi1[ind+imageW+1]-d_phi1[ind+imageW-1])/2;}
		if(dxplus<0){maxdxplus=0;} else { maxdxplus= dxplus*dxplus; }
		if(-dxminus<0){maxminusdxminus=0;} else { maxminusdxminus= dxminus*dxminus; }
		if(dxplus>0){mindxplus=0;} else { mindxplus= dxplus*dxplus; }
		if(-dxminus>0){minminusdxminus=0;} else { minminusdxminus= dxminus*dxminus; }

		if(r==0||r==imageH-1){dy=0;} else {dy=(d_phi1[ind-imageW]-d_phi1[ind+imageW])/2;}
		if(r==0){dyplus=0;} else {dyplus=(d_phi1[ind-imageW]-d_phi1[ind]);}
		if(r==imageH-1){dyminus=0;} else {dyminus=(d_phi1[ind]-d_phi1[ind+imageW]);}
		if(r==0||c==imageW-1||r==imageH-1){dyplusx=0;} else {dyplusx=(d_phi1[ind-imageW+1]-d_phi1[ind+imageW+1])/2;}
		if(r==0||c==0||r==imageH-1){dyminusx=0;} else {dyminusx=(d_phi1[ind-imageW-1]-d_phi1[ind+imageW-1])/2;}
		if(dyplus<0){maxdyplus=0;} else { maxdyplus= dyplus*dyplus; }
		if(-dyminus<0){maxminusdyminus=0;} else { maxminusdyminus= dyminus*dyminus; }
		if(dyplus>0){mindyplus=0;} else { mindyplus= dyplus*dyplus; }
		if(-dyminus>0){minminusdyminus=0;} else { minminusdyminus= dyminus*dyminus; }

		gradphimax=sqrt((sqrt(maxdxplus+maxminusdxminus))*(sqrt(maxdxplus+maxminusdxminus))+(sqrt(maxdyplus+maxminusdyminus))*(sqrt(maxdyplus+maxminusdyminus)));
		gradphimin=sqrt((sqrt(mindxplus+minminusdxminus))*(sqrt(mindxplus+minminusdxminus))+(sqrt(mindyplus+minminusdyminus))*(sqrt(mindyplus+minminusdyminus)));
		nplusx= dxplus / sqrt(1.192092896e-07F + (dxplus*dxplus) + ((dyplusx + dy)*(dyplusx + dy)*0.25) );
		nplusy= dyplus / sqrt(1.192092896e-07F + (dyplus*dyplus) + ((dxplusy + dx)*(dxplusy + dx)*0.25) );
		nminusx= dxminus / sqrt(1.192092896e-07F + (dxminus*dxminus) + ((dyminusx + dy)*(dyminusx + dy)*0.25) );
		nminusy= dyminus / sqrt(1.192092896e-07F + (dyminus*dyminus) + ((dxminusy + dx)*(dxminusy + dx)*0.25) );
		curvature= ((nplusx-nminusx)+(nplusy-nminusy))/2;
		
		F = (ALPHA * d_D[ind]) + ((1-ALPHA) * curvature);
		if(F>0) {gradphi=gradphimax;} else {gradphi=gradphimin;}
		d_phi[ind]=d_phi1[ind] + (DT * F * gradphi);
	
}
}
		



