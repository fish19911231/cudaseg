
#include <hip/hip_runtime.h>

#define ALPHA		 0.00
#define DT			 0.2

__global__ void updatephi( float *d_phi, float *d_phi1, float *d_D, int imageW, int imageH, 
						 float *dx, float *dxplus, float *dxminus, float *dxplusy, float *dxminusy,
						 float *maxdxplus, float *maxminusdxminus, float *mindxplus, float *minminusdxminus,
						 float *dy, float *dyplus, float *dyminus, float *dyplusx, float *dyminusx,
						 float *maxdyplus, float *maxminusdyminus, float *mindyplus, float *minminusdyminus,
						 float *nplusx,float *nplusy,float *nminusx,float *nminusy,float *curvature,
						 float *gradphimax,float *gradphimin, float *F, float *gradphi)

{
	int c= blockIdx.x * blockDim.x + threadIdx.x;
	int r= blockIdx.y * blockDim.y + threadIdx.y;
	int ind= r*imageW+c;

	if(ind<imageW*imageH){

		if(c==0||c==imageW-1){dx[ind]=0;} else {dx[ind]=(d_phi1[ind+1]-d_phi1[ind-1])/2;}
		if(c==imageW-1){dxplus[ind]=0;} else {dxplus[ind]=(d_phi1[ind+1]-d_phi1[ind]);}
		if(c==0){dxminus[ind]=0;} else {dxminus[ind]=(d_phi1[ind]-d_phi1[ind-1]);}
		if(r==0||c==0||c==imageW-1){dxplusy[ind]=0;} else {dxplusy[ind]=(d_phi1[ind-imageW+1]-d_phi1[ind-imageW-1])/2;}
		if(r==imageH-1||c==0||c==imageW-1){dxminusy[ind]=0;} else {dxminusy[ind]=(d_phi1[ind+imageW+1]-d_phi1[ind+imageW-1])/2;}
		if(dxplus[ind]<0){maxdxplus[ind]=0;} else { maxdxplus[ind]= dxplus[ind]*dxplus[ind]; }
		if(-dxminus[ind]<0){maxminusdxminus[ind]=0;} else { maxminusdxminus[ind]= dxminus[ind]*dxminus[ind]; }
		if(dxplus[ind]>0){mindxplus[ind]=0;} else { mindxplus[ind]= dxplus[ind]*dxplus[ind]; }
		if(-dxminus[ind]>0){minminusdxminus[ind]=0;} else { minminusdxminus[ind]= dxminus[ind]*dxminus[ind]; }

		if(r==0||r==imageH-1){dy[ind]=0;} else {dy[ind]=(d_phi1[ind-imageW]-d_phi1[ind+imageW])/2;}
		if(r==0){dyplus[ind]=0;} else {dyplus[ind]=(d_phi1[ind-imageW]-d_phi1[ind]);}
		if(r==imageH-1){dyminus[ind]=0;} else {dyminus[ind]=(d_phi1[ind]-d_phi1[ind+imageW]);}
		if(r==0||c==imageW-1||r==imageH-1){dyplusx[ind]=0;} else {dyplusx[ind]=(d_phi1[ind-imageW+1]-d_phi1[ind+imageW+1])/2;}
		if(r==0||c==0||r==imageH-1){dyminusx[ind]=0;} else {dyminusx[ind]=(d_phi1[ind-imageW-1]-d_phi1[ind+imageW-1])/2;}
		if(dyplus[ind]<0){maxdyplus[ind]=0;} else { maxdyplus[ind]= dyplus[ind]*dyplus[ind]; }
		if(-dyminus[ind]<0){maxminusdyminus[ind]=0;} else { maxminusdyminus[ind]= dyminus[ind]*dyminus[ind]; }
		if(dyplus[ind]>0){mindyplus[ind]=0;} else { mindyplus[ind]= dyplus[ind]*dyplus[ind]; }
		if(-dyminus[ind]>0){minminusdyminus[ind]=0;} else { minminusdyminus[ind]= dyminus[ind]*dyminus[ind]; }

		gradphimax[ind]=sqrt((sqrt(maxdxplus[ind]+maxminusdxminus[ind]))*(sqrt(maxdxplus[ind]+maxminusdxminus[ind]))+(sqrt(maxdyplus[ind]+maxminusdyminus[ind]))*(sqrt(maxdyplus[ind]+maxminusdyminus[ind])));
		gradphimin[ind]=sqrt((sqrt(mindxplus[ind]+minminusdxminus[ind]))*(sqrt(mindxplus[ind]+minminusdxminus[ind]))+(sqrt(mindyplus[ind]+minminusdyminus[ind]))*(sqrt(mindyplus[ind]+minminusdyminus[ind])));
		nplusx[ind]= dxplus[ind] / sqrt(1.192092896e-07F + (dxplus[ind]*dxplus[ind]) + ((dyplusx[ind] + dy[ind])*(dyplusx[ind] + dy[ind])*0.25) );
		nplusy[ind]= dyplus[ind] / sqrt(1.192092896e-07F + (dyplus[ind]*dyplus[ind]) + ((dxplusy[ind] + dx[ind])*(dxplusy[ind] + dx[ind])*0.25) );
		nminusx[ind]= dxminus[ind] / sqrt(1.192092896e-07F + (dxminus[ind]*dxminus[ind]) + ((dyminusx[ind] + dy[ind])*(dyminusx[ind] + dy[ind])*0.25) );
		nminusy[ind]= dyminus[ind] / sqrt(1.192092896e-07F + (dyminus[ind]*dyminus[ind]) + ((dxminusy[ind] + dx[ind])*(dxminusy[ind] + dx[ind])*0.25) );
		curvature[ind]= ((nplusx[ind]-nminusx[ind])+(nplusy[ind]-nminusy[ind]))/2;
		
		F[ind] = (ALPHA * d_D[ind]) + ((1-ALPHA) * curvature[ind]);
		if(F[ind]>0) {gradphi[ind]=gradphimax[ind];} else {gradphi[ind]=gradphimin[ind];}
		d_phi[ind]=d_phi1[ind] + (DT * F[ind] * gradphi[ind]);
	
}
}
		



